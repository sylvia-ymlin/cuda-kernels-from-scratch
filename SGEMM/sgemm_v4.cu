// 2d tiles

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

// one thread computes TM * TN elements
template<const int BM,
        const int BN,
        const int BK,
        const int TM,
        const int TN>
__global__ void sgemm_v4(int M, int N, int K,
    float alpha, float*A, float*B, float beta, float*C){
    
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int block_row_thread = BN / TN;
    int block_col_thread = BM / TM;
    int thread_num = block_row_thread * block_col_thread;

    int tx = (threadIdx.x % block_col_thread) * TN;
    int ty = (threadIdx.x / block_col_thread) * TM;

    // shared memory for the tile
    __shared__ float As[BM*BK];
    __shared__ float Bs[BK*BN];

    // MOVE the pointer to the start of the tile
    A = &A[by * BM * K];
    B = &B[bx * BN];
    C = &C[by * BM * N + bx * BN];

    int a_tile_row = threadIdx.x / BK;
    int a_tile_col = threadIdx.x % BK;
    int a_tile_stride = threadIdx.x / BK;

    int b_tile_row = threadIdx.x / BN;
    int b_tile_col = threadIdx.x % BN;
    int b_tile_stride = threadIdx.x / BN;

    // allocate the registers for the tile
    float tmp[TM][TN] = {0.};
    #pragma unroll
    for(int k = 0; k < K; k += BK){ // loop over blocks
        // load data to shared memory
        // each thread responsible for TM * TN elements
        #pragma unroll
        for(int i = 0; i < BM; i += a_tile_stride){
            As[(a_tile_row + i) * BK + a_tile_col] = A[(a_tile_row + i) * K + a_tile_col];
        }
        #pragma unroll
        for(int i = 0; i < BN; i += b_tile_stride){
            Bs[(b_tile_row + i) * BN + b_tile_col] = B[(b_tile_row + i) * N + b_tile_col];
        }
        __syncthreads();

        A += BK;
        B += BK * N;

        // case to register
        #pragma unroll
        for(int i = 0; i < BK; i++){
            #pragma unroll
            for(int j = 0; j < TM; j++){
                for(int l = 0; l < TN; l++){
                    tmp[j][l] += As[(ty + j) * BK + i] * Bs[i * BN + tx + l ];
                }
            }
        }
        __syncthreads();
    }

    // write the result to the global memory
    #pragma unroll
    for(int j = 0; j < TM; j++){
        for(int l = 0; l < TN; l++){
            C[(ty + j) * N + tx + l] = alpha * tmp[j][l] + beta * C[(ty + j) * N + tx + l];
        }
    }
}

template __global__ void sgemm_v4<128, 128, 8, 8, 8>(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C);